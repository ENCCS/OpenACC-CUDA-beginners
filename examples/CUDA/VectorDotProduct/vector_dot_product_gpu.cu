
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

// Change the code here:
// This should be changed to GPU kernel definition
void dot(int numElements, const float3* a, const float3* b, float* c)
{
    for (int i = 0; i < numElements; i++)
    {
        c[i] = a[i].x*b[i].x + a[i].y*b[i].y + a[i].z*b[i].z;
    }
}

int main()
{
    int numElements = 10000;

    float3* a = (float3*)calloc(numElements, sizeof(float3));
    float3* b = (float3*)calloc(numElements, sizeof(float3));
    float* c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        a[i].x = float(rand())/float(RAND_MAX + 1.0);
        a[i].y = float(rand())/float(RAND_MAX + 1.0);
        a[i].z = float(rand())/float(RAND_MAX + 1.0);

        b[i].x = float(rand())/float(RAND_MAX + 1.0);
        b[i].y = float(rand())/float(RAND_MAX + 1.0);
        b[i].z = float(rand())/float(RAND_MAX + 1.0);
    }

    // Insert your code here:
    // 1. Create GPU device buffers
    // 2. Copy input data from host to device (vectors a and b)
    // 3. Change the CPU function call to the GPU kernel call
    dot(numElements, a, b, c);
    // 4. Copy the result back (vector c)

    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f*%f + %f*%f + %f*%f = %f\n", a[i].x, b[i].x, a[i].y, b[i].y, a[i].z, b[i].z, c[i]);
    }
    printf("...\n");

    free(a);
    free(b);
    free(c);

    // Free GPU memory here
    
    return 0;
}
