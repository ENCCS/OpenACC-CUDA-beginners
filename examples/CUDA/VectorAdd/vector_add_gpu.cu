
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

// Insert your code here:
// GPU kernel definition

int main()
{
    int numElements = 10000;

    float* a = (float*)calloc(numElements, sizeof(float));
    float* b = (float*)calloc(numElements, sizeof(float));
    float* c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        a[i] = float(rand())/float(RAND_MAX + 1.0);
        b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    // Insert your code here:
    // 1. Create GPU device buffers
    // 2. Copy input data from host to device (vectors a and b)
    // 3. Execute the GPU kernel
    // 4. Copy the result back (vector c)
    
    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f + %f = %f\n", a[i], b[i], c[i]);
    }
    printf("...\n");
    
    return 0;
}

