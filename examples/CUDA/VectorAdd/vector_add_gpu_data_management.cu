#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

// Change the code here:
// This should be changed to GPU kernel definition
void vecAdd(int numElements, const float* a, const float* b, float* c)
{
    for (int i = 0; i < numElements; i++)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int numElements = 10000;

    float* h_a = (float*)calloc(numElements, sizeof(float));
    float* h_b = (float*)calloc(numElements, sizeof(float));
    float* h_c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        h_a[i] = float(rand())/float(RAND_MAX + 1.0);
        h_b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        a[i] = float(rand())/float(RAND_MAX + 1.0);
        b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    // Insert your code here:
    
    // 1. Create GPU device buffers
    
    float* d_a;
    float* d_b;
    float* d_c;

    hipMalloc((void**)&d_a, numElements*sizeof(float));
    hipMalloc((void**)&d_b, numElements*sizeof(float));
    hipMalloc((void**)&d_c, numElements*sizeof(float));
    
    // 2. Copy input data from host to device (vectors a and b)
    hipMemcpy(d_a, h_a, numElements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, numElements*sizeof(float), hipMemcpyHostToDevice);

    // 3. Change the CPU function call to the GPU kernel call
    vecAdd(numElements, a, b, c);

    // 4. Copy the result back (vector c)
    hipMemcpy(h_c, d_c, numElements*sizeof(float), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f + %f = %f\n", a[i], b[i], c[i]);
    }
    printf("...\n");

    free(a);
    free(b);
    free(c);
    
    return 0;
}

