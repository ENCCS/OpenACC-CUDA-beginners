
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

#define BLOCK_SIZE 256

__global__ void vecAdd(int numElements, const float* a, const float* b, float* c)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int numElements = 10000;

    float* h_a = (float*)calloc(numElements, sizeof(float));
    float* h_b = (float*)calloc(numElements, sizeof(float));
    float* h_c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        h_a[i] = float(rand())/float(RAND_MAX + 1.0);
        h_b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    float* d_a;
    float* d_b;
    float* d_c;
    
    hipMalloc((void**)&d_a, numElements*sizeof(float));
    hipMalloc((void**)&d_b, numElements*sizeof(float));
    hipMalloc((void**)&d_c, numElements*sizeof(float));

    hipMemcpy(d_a, h_a, numElements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, numElements*sizeof(float), hipMemcpyHostToDevice);

    vecAdd<<<numElements/BLOCK_SIZE + 1, BLOCK_SIZE>>>(numElements, d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, numElements*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }
    printf("...\n");

    free(h_a);
    free(h_b);
    free(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}

