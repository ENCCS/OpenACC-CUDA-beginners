
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

__global__ void vecAdd(int numElements, const float* a, const float* b, float* c)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int numElements = 10000;

    float* a = (float*)calloc(numElements, sizeof(float));
    float* b = (float*)calloc(numElements, sizeof(float));
    float* c = (float*)calloc(numElements, sizeof(float));

    srand(1214134);
    for (int i = 0; i < numElements; i++)
    {
        a[i] = float(rand())/float(RAND_MAX + 1.0);
        b[i] = float(rand())/float(RAND_MAX + 1.0);
    }

    float* d_a;
    float* d_b;
    float* d_c;
    
    hipMalloc((void**)&d_a, numElements*sizeof(float));
    hipMalloc((void**)&d_b, numElements*sizeof(float));
    hipMalloc((void**)&d_c, numElements*sizeof(float));

    hipMemcpy(d_a, a, numElements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, numElements*sizeof(float), hipMemcpyHostToDevice);

    vecAdd<<<numElements/256 + 1, 256>>>(numElements, d_a, d_b, d_c);

    hipMemcpy(c, d_c, numElements*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < std::min(10, numElements); i++)
    {
        printf("%f + %f = %f\n", a[i], b[i], c[i]);
    }
    printf("...\n");
    
    return 0;
}

